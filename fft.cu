#include "hip/hip_runtime.h"
#include<hip/hip_complex.h>
#include<iostream>
#include<math.h>
#include<hip/hip_math_constants.h>
#include <time.h>

//Pode ir

using namespace std;

//Faz exponencial complexa. O parametro � o coeficiente complexo do expoente
__host__ __device__ hipDoubleComplex complexp(double exp) {
  double a = cos(exp);
  double bi = sin(exp);
  return make_hipDoubleComplex(a, bi);
}

__global__ void fft(hipDoubleComplex* A, int m) {
  //Paraleliza a partir do segundo for
  int k = (blockIdx.y*blockDim.y + threadIdx.y)*m;
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  //Assume que a propriedade e^x*e^y = e^(x+y) é valida para 
  //exponencial de numeros complexos
  hipDoubleComplex w = complexp(((2 * HIP_PI) / m)*j);
  hipDoubleComplex t = hipCmul(w, A[k + j + m / 2]);
  hipDoubleComplex u = A[k + j];
  A[k + j] = hipCadd(u, t);
  A[k + j + m / 2] = hipCsub(u, t);
}

//Faz a revers�o de bits do �ndice
__global__ void bit_reverse_copy(hipDoubleComplex* A, int size, hipDoubleComplex* R) {
  int n = blockIdx.x*blockDim.x + threadIdx.x;
  if ( n > size ) return;
  int s = (int)log2((double)size);
  int revn = 0;
  for ( int i = 0; i<s; i++ ) {
    revn += ((n >> i) & 1) << ((s - 1) - i);
  }
  hipDoubleComplex aux = A[n];
  //A[n] = A[revn];
  R[revn] = aux;
}

int main() {
  int p;
  cin >> p;

  int n = (int)pow(2, p);
  int size = n*sizeof(hipDoubleComplex);
  hipDoubleComplex* A = (hipDoubleComplex*)malloc(size);


  for ( int k = 0; k < n; k++ ) {
    A[k].x = k % 100;
    A[k].y = 0;
    /*if ( k < 16 ) {
      A[k].x = 0;
      A[k].y = 0;
    } else {
      A[k].x = 1;
      A[k].y = 0;
    }*/
  }

  clock_t start = clock();
  hipDoubleComplex* A_d, *B_d;
  hipMalloc(&A_d, size);
  hipMalloc(&B_d, size);
  hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);

  int t = (n) > 512 ? 512 : (n);
  dim3 g(t);
  dim3 b((n) / t);

  //cout << "bg: " << g.x << " " << b.x << endl;


  bit_reverse_copy << <g, b >> >(A_d, n, B_d);

  /*hipMemcpy(A, B_d, size, hipMemcpyDeviceToHost);

  for ( int k = 0; k < 32; k++ ) {
    cout <<
      A[k].x
      <<
      " "
      <<
      A[k].y
      <<
      endl;
  }

  cout << "FIM BIT REVERSE" << endl;*/

  /*hipDoubleComplex* B = (hipDoubleComplex*)malloc(size);
  hipMemcpy(B, A_d, size, hipMemcpyDeviceToHost);

  for ( int k = 0; k < 32; k++ ) {
  cout <<
  B[k].x
  <<
  " "
  <<
  B[k].y
  <<
  endl;
  }*/

  int m = 2;
  for ( int i = 1; i <= log2((double)n); i++ ) {
    //Divide o trabalho proporcionalmente
    int nk = n / m;
    int nj = m / 2;
    int num = nj* nk;
    double prop = ((double)nk) / ((double)num);
    int threads = num > 512 ? 512 : num;
    int py = (int)(threads*prop);
    //Trata o caso de a propor��o de trabalho de k ser 
    //tão pequena que d menos que uma thread por bloco
    int y = (py >= 1) ? py : 1;
    int x = threads / y;
    int by = nk / y;
    int bx = nj / x;
    //cout << "Elas sao: " << x << " " << y << " " << bx << " " << by << endl;
    dim3 grid(x, y);
    dim3 blocks(bx, by);
    //cout << "Chamei fft:" << endl;
    fft << <grid, blocks >> >(B_d, m);
    //cout << i << " " << hipGetErrorString(hipGetLastError()) << endl;
    //cout << "Sai do fft:" << endl;
    m *= 2;
  }
  hipMemcpy(A, B_d, size, hipMemcpyDeviceToHost);

  hipFree(A_d);
  hipFree(B_d);
  clock_t end = clock();
  float sec = (float)(end - start) / CLOCKS_PER_SEC;
  cout << sec << " seconds elapsed!" << endl;

  for ( int k = 0; k < 32; k++ ) {
    cout <<
      A[k].x
      <<
      " "
      <<
      A[k].y
      <<
      endl;
  }
  free(A);

  cout << "Acabei o procedimento..." << endl;
  int d;
  cin >> d;
}



